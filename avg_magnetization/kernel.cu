#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cmath>
#include <cstring>  
#include <string>
#include <sstream>  
#include <thrust/reduce.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include "cu_veclib.cuh"

#define PI = 3.14159

using namespace std;


__global__ void init_map(vec *d_coord){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	if (i < 128 && j < 128){
		d_coord[i + j * 128].x = i*0.001 / (double)128;
		d_coord[i + j * 128].y = j*0.001 / (double)128;
		d_coord[i + j * 128].z = 0;
	}
}

__global__ void init_pos(vec *d_pil_pos, double x_off, double y_off){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < 50){
		d_pil_pos[i].x = d_pil_pos[i].x + x_off;
		d_pil_pos[i].y = d_pil_pos[i].y + x_off;
	}
}

__global__ void init_dist(vec *d_pil_pos, vec *d_coord, vec *d_dist){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	if (i < 128*128 && j < 50){
		d_dist[i + 128 * 128 * j] = d_coord[i + 128 * 128 * j]-d_pil_pos[j];
	}
}
__global__ void calc_H(vec *d_dist, vec *d_dip, vec *d_Hi_inc, vec *d_Hi_tot, vec *d_Hi_temp, int *d_keys){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	if (i < 128 * 128 && j < 50){
		d_Hi_inc[i + 128 * 128 * j] = (double)3 * d_dist[i + 128 * 128 * j] * (d_dip[i + 128 * 128 * j] * d_dist[i + 128 * 128 * j]) / pow(d_dist[i + 128 * 128 * j].abs(), 5) - d_dip[i + 128 * 128 * j] / pow(d_dist[i + 128 * 128 * j].abs(),3);
		d_Hi_temp[j + 128 * 128 * i] = d_Hi_inc[i + 128 * 128 * j];
	}
	if (i < 128 * 128 * 50){
		d_keys[i] = 50;
	}
}


int main(){
	// PLACA GR�FICA: NVIDIA GEFORCE 820M - COMPUTE CAPABILITY: 2.1

	int ndip = 50; //# de dipolos
	int ni = 128; //# de threads (linhas da matriz)
	int nj = 128*ndip; //# de blocos (linhas da matriz*dipolos do pilar = 6400)
	int i, j = 0, k = 0, l = 0;
	bool a = 1;
	string fileplace_def, fileplace_vec;

	int threadsPerBlock = ni;
	int BlocksPerGrid = nj;
	double x_off=0.0005, y_off=0.0005;

	vec *h_dip, *h_pil_pos, *h_Hi_inc, *temp_dip;
	vec h_Hi_avg(0, 0, 0);


	//---------------------------ALOCA��O DE ESPA�O----------------------------//
	h_dip = (vec*)malloc(sizeof(vec)*ndip); //Vector de magnetiza��o dos dipolos
	temp_dip = (vec*)malloc(sizeof(vec)*10); //Vector auxiliar
	h_pil_pos = (vec*)malloc(sizeof(vec)*ndip); //Posi��o dos dipolos
	h_Hi_inc = (vec*)malloc(sizeof(vec)*ni*ni); //Valor do campo incidente num elemento de �rea 
	
	int *d_keys, *d_rest;
	vec *d_dist, *d_pil_pos, *d_Hi_inc, *d_dip, *d_coord, *d_Hi_temp, *d_Hi_tot;
	vec *d_Mi_avg_p;

	hipMalloc(&d_Mi_avg_p, sizeof(vec));
	hipMalloc(&d_dist, sizeof(vec)*ni*nj);
	hipMalloc(&d_keys, sizeof(int)*ni*nj);
	hipMalloc(&d_rest, sizeof(int));
	hipMalloc(&d_pil_pos, sizeof(vec)*ndip);
	hipMalloc(&d_Hi_inc, sizeof(vec)*ni*nj);
	hipMalloc(&d_Hi_tot, sizeof(vec)*ni*ni);
	hipMalloc(&d_Hi_temp, sizeof(vec)*ni*nj);
	hipMalloc(&d_dip, sizeof(vec)*ndip);
	hipMalloc(&d_coord, sizeof(vec)*ni*ni);

	cout << "POWERED BY CUDA" << endl << endl;

	_sleep(2000);

	for (i = 1; i < 2; i++){

		//------- PARSING (Entrada de dados) -------//

		// O delimitador � qq caracter que n�o seja um n�mero v�lido (para ints e floating points de qq precis�o)

		fileplace_def = "C:\\Users\\Pedro\\Documents\\MATLAB\\dados\\defs_"+to_string(i)+".txt";
		fileplace_vec = "C:\\Users\\Pedro\\Documents\\MATLAB\\dados\\vecs_"+to_string(i)+".txt";

		fstream filein_def;
		fstream filein_vec;
		filein_def.open(fileplace_def);
		filein_vec.open(fileplace_vec);

		for (j = 0; j < 10; j++){
			filein_vec >> temp_dip[j].x >> temp_dip[j].y >> temp_dip[j].z;
		}

		filein_vec.close();

		for (j = 0; j < ndip; j++){
			k++;
			if (k == 10)
				k = 0;
			filein_def >> h_pil_pos[j].x >> h_pil_pos[j].y >> h_pil_pos[j].z;
			h_dip[j] = temp_dip[k];
		}

		filein_def.close();

		//------ DATA TRANSFER (H -> D) --------//

		hipMemcpy(d_dip, h_dip, sizeof(vec)*ndip, hipMemcpyHostToDevice);
		hipMemcpy(d_pil_pos, h_pil_pos, sizeof(vec)*ndip, hipMemcpyHostToDevice);

		//-----------INICIALIZA��O------------------//

		init_map << <BlocksPerGrid, threadsPerBlock >> >(d_coord);
		init_pos << <BlocksPerGrid, threadsPerBlock >> >(d_pil_pos, x_off, y_off);
		init_dist << <BlocksPerGrid, threadsPerBlock >> >(d_pil_pos, d_coord, d_dist);

		//------ C�LCULO ------ //
		calc_H << <BlocksPerGrid, threadsPerBlock >> >(d_dist, d_dip, d_Hi_inc, d_Hi_tot, d_Hi_temp, d_keys);
		thrust::device_ptr<vec> d_Hi_temp_thrust = thrust::device_pointer_cast(d_Hi_temp);
		thrust::device_ptr<vec> d_Hi_tot_thrust = thrust::device_pointer_cast(d_Hi_tot);
		thrust::device_ptr<int> d_keys_thrust = thrust::device_pointer_cast(d_keys);
		thrust::device_ptr<int> d_rest_thrust = thrust::device_pointer_cast(d_rest);
		thrust::reduce_by_key(d_keys_thrust, d_keys_thrust + 128 * 128, d_Hi_temp_thrust, d_rest_thrust, d_Hi_tot_thrust);
		
		//for (i = 0; i < ni*nj; i++){
		//	Mi_avg = Mi_avg + h_Mi[i];
		//}
		//Mi_avg = Mi_avg*((double)1 / ((ni - 2)*(nj - 2)));

		//Mi_avg_p = Mi_avg;
		//Mi_avg = Ms*Mi_avg.norm();

		//hipMemcpy(d_Mi_avg, &Mi_avg, sizeof(vec), hipMemcpyHostToDevice);

		//define_Mi << <BlocksPerGrid, threadsPerBlock >> >(d_Mi, d_Mi_avg);
		//d_rand << <BlocksPerGrid, threadsPerBlock >> >(d_Mi, s, theta, r);
		//stripes << <BlocksPerGrid, threadsPerBlock >> >(d_Mi);




		//------ DATA TRANSFER (D -> H) --------//


		//hipDeviceSynchronize();

		////hipMemcpy(d_H, &h_H, sizeof(vec), hipMemcpyHostToDevice);
		//HMj << <BlocksPerGrid, threadsPerBlock >> >(d_Hmj, h_H, d_Mi);
		//kMj << <BlocksPerGrid, threadsPerBlock >> >(d_kMj, d_Mi);
		//exch << <BlocksPerGrid, threadsPerBlock >> >(d_exch, d_Mi);
		//demag << <BlocksPerGrid, threadsPerBlock >> >(d_demag, d_Mi);
		//energy << <BlocksPerGrid, threadsPerBlock >> >(d_Ej, d_Hmj, d_kMj, d_exch, d_demag);

		//thrust::device_ptr<double> d_energy_thrust = thrust::device_pointer_cast(d_Ej);

		//Et = thrust::reduce(d_energy_thrust, d_energy_thrust + ni*nj);

		//Et_p = Et;

		//cout << "Introduzir valor de Hx" << endl;
		//cin >> h_H.x;
		//cout << "Introduzir valor de Hy" << endl;
		//cin >> h_H.y;

		//cout << "Foi introduzido Hx=" << h_H.x << " e Hy=" << h_H.y << endl;

		//cout << "Introduza o nome do ficheiro" << endl;
		//cin >> nficheiro;

		//_sleep(2000);

		//ofstream tofile(nficheiro);
		////ofstream tofile2("Mi_x_2_d_term.txt");
		//srand(time(NULL));

		//k = 0;
		//Et_p = 1e10;
		//Et = 1e11;
		////hipMemcpy(d_H, &h_H, sizeof(vec), hipMemcpyHostToDevice);
		//cout << h_H.x << endl;
		//while (k != 10){
		//	while (false == thermal(Et_p, Et, kB, Vcell)/*Et_p<=Et*/){
		//		l++;
		//		hipDeviceSynchronize();
		//		d_rand << <BlocksPerGrid, threadsPerBlock >> >(d_Mi, s, theta, r);
		//		stripes << <BlocksPerGrid, threadsPerBlock >> >(d_Mi);
		//		HMj << <BlocksPerGrid, threadsPerBlock >> >(d_Hmj, h_H, d_Mi);
		//		kMj << <BlocksPerGrid, threadsPerBlock >> >(d_kMj, d_Mi);
		//		exch << <BlocksPerGrid, threadsPerBlock >> >(d_exch, d_Mi);
		//		demag << <BlocksPerGrid, threadsPerBlock >> >(d_demag, d_Mi);
		//		energy << <BlocksPerGrid, threadsPerBlock >> >(d_Ej, d_Hmj, d_kMj, d_exch, d_demag);

		//		Et = thrust::reduce(d_energy_thrust, d_energy_thrust + ni*nj);

		//		if (l == 1000){
		//			//cout << "Nao convergiu :(" << endl;
		//			a = false;
		//			break;
		//		}
		//		a = 1;
		//	}
		//	if (a == 1){
		//		Et_p = Et;
		//		hipMemcpy(h_Mi, d_Mi, sizeof(vec)*ni*nj, hipMemcpyDeviceToHost);
		//		h_Mi_p = h_Mi;

		//		for (i = 0; i < ni*nj; i++){
		//			Mi_avg = Mi_avg + h_Mi[i];
		//		}

		//		Mi_avg = Mi_avg*((double)1 / ((ni - 2)*(nj - 2)));

		//		if (abs(Mi_avg.x - Mi_avg_p.x) < .05 && abs(Mi_avg.y - Mi_avg_p.y) < .05)
		//			break;

		//		Mi_avg_p = Mi_avg;
		//		Mi_avg = Ms*Mi_avg_p.norm();

		//		hipMemcpy(d_Mi_avg, &Mi_avg, sizeof(vec), hipMemcpyHostToDevice);
		//		define_Mi << <BlocksPerGrid, threadsPerBlock >> >(d_Mi, d_Mi_avg);
		//		//hipMemcpy(h_Mi, d_Mi, sizeof(vec)*ni*nj, hipMemcpyDeviceToHost);
		//		k = 0;
		//		cout << Mi_avg << endl;
		//	}
		//	else{
		//		k++;
		//		cout << k << endl;
		//		hipMemcpy(d_Mi, h_Mi, sizeof(vec)*ni*nj, hipMemcpyHostToDevice);
		//	}
		//	hipDeviceSynchronize();
		//	l = 0;
		//	//hipMemcpy(d_Mi, h_Mi, sizeof(vec)*ni*nj, hipMemcpyHostToDevice);
		//	Mi_avg.x = 0;
		//	Mi_avg.y = 0;
		//	Mi_avg.z = 0;
		//}
		//tofile << h_H.x << "\t" << Mi_avg_p << endl;
		//hipMemcpy(h_Mi, d_Mi, sizeof(vec)*ni*nj, hipMemcpyDeviceToHost);
		//hipMemcpy(h_Hmj, d_Hmj, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);
		//hipMemcpy(h_kMj, d_kMj, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);
		//hipMemcpy(h_exch, d_exch, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);
		//hipMemcpy(h_demag, d_demag, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);
		//hipMemcpy(h_Ej, d_Ej, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);


		////init_rand << <BlocksPerGrid, threadsPerBlock >> >(s);
		////d_rand << <BlocksPerGrid, threadsPerBlock >> >(teste, s);

		////init_Mi << <BlocksPerGrid, threadsPerBlock >> >(theta, ni, nj);


		//hipMemcpy(h_Hmj, d_Hmj, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);
		//for (i = 0; i < nj; i++){
		//	for (j = 0; j < ni; j++)
		//		tofile << i << "\t" << j << "\t" << h_Mi_p[j + i*ni] << endl;
		//}

		//tofile.close();

		//hipMemcpy(teste1, teste, sizeof(double)*ni*nj, hipMemcpyDeviceToHost);

	}


	//----------------------------Liberta��o de espa�o-----------------------------//
	
	hipFree(d_Mi_avg_p);
	hipFree(d_dist);
	hipFree(d_pil_pos);
	hipFree(d_Hi_inc);
	hipFree(d_dip);
	hipFree(d_coord);

	free(h_dip);
	free(h_pil_pos);
	free(h_Hi_inc);
	
	cout << "PROGRAMA CORRIDO COM SUCESSO!" << endl << "Prima qualquer tecla para sair..." << endl;
	cin.get();
	return 0;
}
