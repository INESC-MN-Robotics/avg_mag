#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cmath>
#include <cstring>  
#include <string>
#include <sstream>  
#include <thrust/reduce.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include "cu_veclib.cuh"

#define PI = 3.14159

using namespace std;


__global__ void init_map(vec *d_coord){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	if (i < 128 && j < 128){
		d_coord[i + j * 128].x = i*0.001 / (double)128;
		d_coord[i + j * 128].y = j*0.001 / (double)128;
		d_coord[i + j * 128].z = 0;
	}
}

__global__ void init_pos(vec *d_pil_pos, double x_off, double y_off){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < 50){
		d_pil_pos[i].x = d_pil_pos[i].x + x_off;
		d_pil_pos[i].y = d_pil_pos[i].y + x_off;
	}
}

__global__ void init_dist(vec *d_pil_pos, vec *d_coord, vec *d_dist){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	int k;
	if (i < 128 && j < 128){
		for (k = 0; k < 50; k++)
			d_dist[k + 50 * i + 50 * 128 * j] = d_coord[i + 128 * j] - d_pil_pos[k];
	}
}
__global__ void calc_H(vec *d_dist, vec *d_dip, vec *d_Hi_inc, vec *d_Hi_tot, int *d_keys){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int j = blockDim.y*blockIdx.y + threadIdx.y;
	int k;
	if (i < 128 && j < 128){
		for (k = 0; k < 50; k++)
			d_Hi_inc[k + 50 * i + 50 * 128 * j] = (double)3 * d_dist[k + 50 * i + 50 * 128 * j] * (d_dip[k] * d_dist[k + 50 * i + 50 * 128 * j]) / pow(d_dist[k + 50 * i + 50 * 128 * j].abs(), 5) - d_dip[k] / pow(d_dist[k + 50 * i + 50 * 128 * j].abs(), 3);
	}
	if (i < 128 * 128 * 50){
		d_keys[i] = 50;
	}
}


int main(){
	// PLACA GR�FICA: NVIDIA GEFORCE 820M - COMPUTE CAPABILITY: 2.1

	int ndip = 50; //# de dipolos
	int ni = 128; //# de threads (linhas da matriz)
	int nj = 128*ndip; //# de blocos (linhas da matriz*dipolos do pilar = 6400)
	int i, j = 0, k = 0;
	string fileplace_def, fileplace_vec, fileplace_points, fileplace_avgs;

	int threadsPerBlock = ni;
	int BlocksPerGrid = nj;
	double x_off=0.0005, y_off=0.0005;

	vec *h_dip, *h_pil_pos, *h_Hi_inc, *temp_dip;
	vec h_Hi_avg(0, 0, 0);


	//---------------------------ALOCA��O DE ESPA�O----------------------------//
	h_dip = (vec*)malloc(sizeof(vec)*ndip); //Vector de magnetiza��o dos dipolos
	temp_dip = (vec*)malloc(sizeof(vec)*10); //Vector auxiliar
	h_pil_pos = (vec*)malloc(sizeof(vec)*ndip); //Posi��o dos dipolos
	h_Hi_inc = (vec*)malloc(sizeof(vec)*ni*ni); //Valor do campo incidente num elemento de �rea 
	
	int *d_keys, *d_rest;
	vec *d_dist, *d_pil_pos, *d_Hi_inc, *d_dip, *d_coord, *d_Hi_tot;
	vec *d_Hi_avg;

	hipMalloc(&d_Hi_avg, sizeof(vec));
	hipMalloc(&d_dist, sizeof(vec)*ni*nj);
	hipMalloc(&d_keys, sizeof(int)*ni*nj);
	hipMalloc(&d_rest, sizeof(int));
	hipMalloc(&d_pil_pos, sizeof(vec)*ndip);
	hipMalloc(&d_Hi_inc, sizeof(vec)*ni*nj);
	hipMalloc(&d_Hi_tot, sizeof(vec)*ni*ni);
	hipMalloc(&d_dip, sizeof(vec)*ndip);
	hipMalloc(&d_coord, sizeof(vec)*ni*ni);

	thrust::device_ptr<vec> Hi_tot_thrust = thrust::device_pointer_cast(d_Hi_tot);
	thrust::device_ptr<vec> Hi_inc_thrust = thrust::device_pointer_cast(d_Hi_inc);
	thrust::device_ptr<int> keys_thrust = thrust::device_pointer_cast(d_keys);
	thrust::device_ptr<int> rest_thrust = thrust::device_pointer_cast(d_rest);

	cout << "POWERED BY CUDA" << endl << endl;

	_sleep(2000);

	fstream fileout_avgs;
	fileplace_avgs = "C:\\Users\\Pedro\\Documents\\MATLAB\\dados\\avgs.txt";
	fileout_avgs.open(fileplace_avgs);

	for (i = 1; i < 2; i++){

		h_Hi_avg.x = 0;
		h_Hi_avg.y = 0;
		h_Hi_avg.z = 0;

		//------- PARSING (Entrada de dados) -------//

		// O delimitador � qq caracter que n�o seja um n�mero v�lido (para ints e floating points de qq precis�o)

		fileplace_def = "C:\\Users\\Pedro\\Documents\\MATLAB\\dados\\defs_"+to_string(i)+".txt";
		fileplace_vec = "C:\\Users\\Pedro\\Documents\\MATLAB\\dados\\vecs_"+to_string(i)+".txt";

		fstream filein_def;
		fstream filein_vec;
		filein_def.open(fileplace_def);
		filein_vec.open(fileplace_vec);

		for (j = 0; j < 10; j++){
			filein_vec >> temp_dip[j].x >> temp_dip[j].y >> temp_dip[j].z;
		}

		filein_vec.close();

		for (j = 0; j < ndip; j++){
			k++;
			if (k == 10)
				k = 0;
			filein_def >> h_pil_pos[j].x >> h_pil_pos[j].y >> h_pil_pos[j].z;
			h_dip[j] = temp_dip[k];
		}

		filein_def.close();

		//------ DATA TRANSFER (H -> D) --------//

		hipMemcpy(d_dip, h_dip, sizeof(vec)*ndip, hipMemcpyHostToDevice);
		hipMemcpy(d_pil_pos, h_pil_pos, sizeof(vec)*ndip, hipMemcpyHostToDevice);

		//-----------INICIALIZA��O------------------//

		init_map << <BlocksPerGrid, threadsPerBlock >> >(d_coord);
		init_pos << <BlocksPerGrid, threadsPerBlock >> >(d_pil_pos, x_off, y_off);
		init_dist << <BlocksPerGrid, threadsPerBlock >> >(d_pil_pos, d_coord, d_dist);

		//------ C�LCULO ------ //
		calc_H << <BlocksPerGrid, threadsPerBlock >> >(d_dist, d_dip, d_Hi_inc, d_Hi_tot, d_keys);
		thrust::reduce_by_key(keys_thrust, keys_thrust + ni * nj, Hi_inc_thrust, rest_thrust, Hi_tot_thrust);

		//------ DATA TRANSFER (D -> H) --------//

		//hipMemcpy(h_dist, d_Hi_inc, sizeof(vec)*ni*nj, hipMemcpyDeviceToHost); //UNCOMMENT TO DIAGNOSE
		fstream fileout_points;
		fileplace_points = "C:\\Users\\Pedro\\Documents\\CUDA_OUT\\points_" + to_string(i) + ".txt";
		fileout_points.open(fileplace_points);
		for (i = 0; i < ni; i++){
			for (j = 0; j < ni; j++){
				fileout_points << Hi_tot_thrust[i];
				h_Hi_avg = h_Hi_avg + Hi_tot_thrust[i];
			}
			fileout_points << endl;
		}
		h_Hi_avg = h_Hi_avg / (128 * 128);
		fileout_points.close();
		fileout_avgs << h_Hi_avg << endl;
	}

	fileout_avgs.close();

	//----------------------------Liberta��o de espa�o-----------------------------//
	
	hipFree(d_Hi_avg);
	hipFree(d_dist);
	hipFree(d_pil_pos);
	hipFree(d_Hi_inc);
	hipFree(d_dip);
	hipFree(d_coord);
	hipFree(d_keys);
	hipFree(d_rest);
	hipFree(d_Hi_tot);

	free(h_dip);
	free(h_pil_pos);
	free(h_Hi_inc);
	free(temp_dip);
	
	cout << "PROGRAMA CORRIDO COM SUCESSO!" << endl << "Prima qualquer tecla para sair..." << endl;
	cin.get();
	return 0;
}
